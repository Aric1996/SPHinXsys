#include "hip/hip_runtime.h"
/*
 * Massoud Rezavand 2019.
 * Technical University of Munich
 * gpuSPHinxsys - An SPH solver for CUDA enabled GPUs
 *
 * This is the main interface to communicate between
 * Host and Device
 */

#include"gpuSPHinxsys.cuh"
#include"System.h"
#include"ParticleData.cuh"
#include"ParticleGroup.cuh"
#include"CellList.cuh"
#include"timeStepping.cuh"
#include"dtSizeCalc.cuh"


using gpu::access;
using real = gpu::real;
using real3 = gpu::real3;
using real4 = gpu::real4;
using timeStepping = gpu::timeStepping;


gpuSPHinxsys::gpuSPHinxsys(Parameters par):
    particle_size(par.particle_size),
    box_size(par.box_size),
    body_force(par.body_force),
    probe1(par.probe1), probe2(par.probe2),
    U_f(par.U_f), c_f(par.c_f), rho0_f(par.rho0_f),
    c_s(par.c_s), rho0_s(par.rho0_s), rho0_g(par.rho0_g)
{
    std::cout << "\t*************************************************" << "\n";
    std::cout << "\t       gpuSPHinXsys: An SPH solver on GPUs       " << "\n";
    std::cout << "\t*************************************************" << "\n";
    //    /*checkCudaErrors*/(hipMalloc((void **)&devicePar->VelMax, sizeof(real)));
}

gpuSPHinxsys::~gpuSPHinxsys()
{
    std::cout << "\t*************************************************" << "\n";
    std::cout << "\t             gpuSPHinXsys call ended!            " << "\n";
    std::cout << "\t*************************************************" << "\n";
    //    if(devicePar->VelMax != NULL) hipFree(devicePar->VelMax);
}

struct GPU{
    std::shared_ptr<gpu::ParticleData> pd;
    std::shared_ptr<gpu::System> sys;
    std::shared_ptr<gpu::CellList> nl; //neighborlist
};

//initialize a GPU system
GPU initializeGPU(size_t np){
    auto sys = std::make_shared<gpu::System>();
    auto  pd = std::make_shared<gpu::ParticleData>(np, sys);
    auto  pg = std::make_shared<gpu::ParticleGroup>(pd, sys, "All");
    auto  nl = std::make_shared<gpu::CellList>(pd, pg, sys);
    return {pd, sys, nl};
}

//Copy particle position_type to device and allocate density and mass
void copyPositionsToDevice(const GPU & gpu_state,
                           const std::vector<float> &fluidParticles,
                           const std::vector<float> &wallParticles,
                           const std::vector<float> &thirdBodyParticles,
                           float rho0_fluid,
                           float rho0_gas,
                           int DIM,
                           float dp){
    auto  pos_type = gpu_state.pd->getPos(access::location::cpu, access::mode::write);
    auto       rho = gpu_state.pd->getRho(access::location::cpu, access::mode::write);
    auto      rho0 = gpu_state.pd->getRho0(access::location::cpu, access::mode::write);
    auto      mass = gpu_state.pd->getMass(access::location::cpu, access::mode::write);
    auto       vol = gpu_state.pd->getVol(access::location::cpu, access::mode::write);
    size_t npFluid = fluidParticles.size()/3;
    size_t  npWall = wallParticles.size()/3;
    size_t  npThirdBody = thirdBodyParticles.size()/3;

    for(int i = 0; i<npFluid; i++){
        pos_type[i].x = fluidParticles[3*i];
        pos_type[i].y = fluidParticles[3*i+1];
        pos_type[i].z = fluidParticles[3*i+2];
        pos_type[i].w = LIQUID; //fluid particles have type 0
        rho[i] = rho0_fluid;
        rho0[i] = rho0_fluid;
        mass[i] = std::pow(dp, DIM)*rho[i];
        vol[i] = mass[i]/rho[i];
        // check NaN in fluid pos
        if (std::isnan(pos_type[i].x) or
            std::isnan(pos_type[i].y) or
            std::isnan(pos_type[i].z)){
            std::cout << "ERROR in Fluid position copied to Device!" <<"\n";
            exit(1);
        }
    }
    for(int i = 0; i<npWall; i++){
        pos_type[i+npFluid].x = wallParticles[3*i];
        pos_type[i+npFluid].y = wallParticles[3*i+1];
        pos_type[i+npFluid].z = wallParticles[3*i+2];
        pos_type[i+npFluid].w = WALL; //wall particles have type 1
        rho[i+npFluid] = rho0_fluid;
        rho0[i+npFluid] = rho0_fluid;
        mass[i+npFluid] = std::pow(dp, DIM)*rho[i+npFluid];
        vol[i+npFluid] = mass[i+npFluid]/rho[i+npFluid];
        // check NaN in wall pos
        if (std::isnan(pos_type[i+npFluid].x) or
            std::isnan(pos_type[i+npFluid].y) or
            std::isnan(pos_type[i+npFluid].z)){
            std::cout << "ERROR in Wall position copied to Device!" <<"\n";
            exit(1);
        }
    }
    for(int i = 0; i<npThirdBody; i++){
        pos_type[i+npFluid+npWall].x = thirdBodyParticles[3*i];
        pos_type[i+npFluid+npWall].y = thirdBodyParticles[3*i+1];
        pos_type[i+npFluid+npWall].z = thirdBodyParticles[3*i+2];
        pos_type[i+npFluid+npWall].w = THIRDBODY; //3rd body particles have type 2
        rho[i+npFluid+npWall] = rho0_gas;
        rho0[i+npFluid+npWall] = rho0_gas;
        mass[i+npFluid+npWall] = std::pow(dp, DIM)*rho[i+npFluid+npWall];
        vol[i+npFluid+npWall] = mass[i+npFluid+npWall]/rho[i+npFluid+npWall];
        // check NaN in THIRDBODY pos
        if (std::isnan(pos_type[i+npFluid+npWall].x) or
            std::isnan(pos_type[i+npFluid+npWall].y) or
            std::isnan(pos_type[i+npFluid+npWall].z)){
            std::cout << "ERROR in Thirdbody position copied to Device!" <<"\n";
            exit(1);
        }
    }
}


// set initial velocity and pressure to zero
void initialize_properties(const GPU & gpu_state){
    auto vel = gpu_state.pd->getVel(access::location::cpu, access::mode::write);
    auto   p = gpu_state.pd->getPressure(access::location::cpu, access::mode::write);
    std::fill(vel.begin(), vel.end(), real3());
    std::fill(p.begin(), p.end(), real());
#ifdef _TRANSPORT_VELOCITY_
    auto vel_tv = gpu_state.pd->getVel_tv(access::location::cpu, access::mode::write);
    auto   F_Pb = gpu_state.pd->getF_Pb(access::location::cpu, access::mode::write);
    std::fill(vel_tv.begin(), vel_tv.end(), real3());
    std::fill(F_Pb.begin(), F_Pb.end(), real3());
#endif
}

void gpuSPHinxsys::call_gpuSPHinxsys(std::vector<float> &fluidParticles,
                                     std::vector<float> &wallParticles,
                                     std::vector<float> &thirdBodyParticles){

    int     npFluid = fluidParticles.size()/3;
    int      npWall = wallParticles.size()/3;
    int npThirdBody = thirdBodyParticles.size()/3;
    int          np = npFluid + npWall + npThirdBody;

    printf("Copied to device: npFluid: %d npWall: %d npThirdBody: %d np: %d \n", npFluid, npWall, npThirdBody, np);

    //copy the box size
    real3 boxSize = gpu::make_real3(std::get<0>(box_size),
                                    std::get<1>(box_size),
                                    std::get<2>(box_size));
    //copy the external body force
    real3 bodyForceTmp = gpu::make_real3(std::get<0>(body_force),
                                          std::get<1>(body_force),
                                          std::get<2>(body_force));

    int DIM = boxSize.z > real(0.0)?3:2;

    auto gpu_state = initializeGPU(np);

    copyPositionsToDevice(gpu_state, fluidParticles, wallParticles,
                          thirdBodyParticles, rho0_f, rho0_g, DIM, particle_size);

    //creat a pointer to the timeStepping module
    timeStepping::Parameters parTS;
    parTS.U_f = U_f;
    parTS.box = gpu::Box(boxSize);
    parTS.h = 1.3*particle_size;
    parTS.c_f = c_f;
    parTS.rho0_f = rho0_f;
    parTS.bodyForce = bodyForceTmp;
    parTS.probe1 = probe1;
    parTS.probe2 = probe2;
    auto timeStepping_ptr = std::make_shared<timeStepping>(gpu_state.pd, gpu_state.nl, gpu_state.sys, parTS);

    initialize_properties(gpu_state);

    gpu_state.pd->sortParticles();

    //the time integration functions to Run the simulation
    timeStepping_ptr->velVerletIntg(); //velocity Verlet scheme
//    timeStepping_ptr->dualCriteriaIntg(); //dual criteria scheme
//    timeStepping_ptr->solidDynaIntg(); //integration scheme for solid dynamics

    gpu_state.sys->finish();
}
