#include "hip/hip_runtime.h"
/*
 * Massoud Rezavand 2019.
 * Technical University of Munich
 * gpuSPHinxsys - An SPH solver for CUDA enabled GPUs
 *
 * This module integrates the particles dynamics using
 * different time marching algorithms
 */

#include"timeStepping.cuh"
#include"dtSizeCalc.cuh"
#include"fluidDynamics.cuh"
#include"inOut.cuh"
#include <chrono>


namespace gpu{

timeStepping::timeStepping(shared_ptr<ParticleData> pd,
                           shared_ptr<CellList> nl,
                           shared_ptr<System> sys,
                           timeStepping::Parameters par):
    pd(pd), nl(nl), sys(sys), U_f(par.U_f),
    box(par.box), h(par.h), c_f(par.c_f), probe1(par.probe1),
    probe2(par.probe2), rho0_f(par.rho0_f), bodyForce(par.bodyForce){
    printf("|timeStepping| \tis called with c_f = %.1f, rho0_f = %.1f \n", c_f, rho0_f);
    CudaSafeCall(hipStreamCreate(&stream));
}

timeStepping::~timeStepping(){
    hipStreamDestroy(stream);
    printf("|timeStepping| \tcall ended! \n");
}

namespace timeStepping_ns{
//Kernel for time stepping
template<int halfSteps>
__global__ void integration_ker(real4* __restrict__ pos,
                                real3* __restrict__ vel,
                                real4* __restrict__ force,
                                real3* __restrict__ F_Pb,
                                real3* __restrict__ vel_tv,
                                const int* __restrict__ groupIndex,
                                int np,
                                real dt){
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    if(i>=np) return;

    vel[groupIndex[i]] += make_real3(force[groupIndex[i]])*dt*real(0.5);

    //updat positions at the first half step
    if(halfSteps==timeStepping::firstHalf){
#ifndef _TRANSPORT_VELOCITY_
        real3 newPos = make_real3(pos[groupIndex[i]]) + vel[groupIndex[i]]*dt;
        pos[groupIndex[i]] = make_real4(newPos, pos[groupIndex[i]].w);
#else
        vel_tv[groupIndex[i]] = vel[groupIndex[i]] + F_Pb[groupIndex[i]]*dt*real(0.5);
        real3 newPos = make_real3(pos[groupIndex[i]]) + vel_tv[groupIndex[i]]*dt;
        pos[groupIndex[i]] = make_real4(newPos, pos[groupIndex[i]].w);
        F_Pb[groupIndex[i]] = make_real3(0);
#endif
        //Reset force
        force[groupIndex[i]] = make_real4(0);
    }
}

}//timeStepping_ns

//function for time integration
template<int halfSteps>
void timeStepping::integration(real currentDt){
    int np = pd->getNumParticles();
    int Nthreads=128;
    int Nblocks=np/Nthreads + ((np%Nthreads)?1:0);
    auto groupIndex = nl->getGroupIndexIterator();
    auto   pos = pd->getPos(access::location::gpu, access::mode::readwrite);
    auto   vel = pd->getVel(access::location::gpu, access::mode::readwrite);
    auto force = pd->getForce(access::location::gpu, access::mode::readwrite);
#ifdef _TRANSPORT_VELOCITY_
    auto   F_Pb = pd->getF_Pb(access::location::gpu, access::mode::readwrite).raw();
    auto vel_tv = pd->getVel_tv(access::location::gpu, access::mode::readwrite).raw();
#else
    auto   F_Pb = nullptr;
    auto vel_tv = nullptr;
#endif
    timeStepping_ns::integration_ker<halfSteps><<<Nblocks, Nthreads, 0, stream>>>(pos.raw(),
                                                                                  vel.raw(),
                                                                                  force.raw(),
                                                                                  F_Pb,
                                                                                  vel_tv,
                                                                                  groupIndex,
                                                                                  np,
                                                                                  currentDt);
}

//velocity verlet integration
//called from the main inerface to run the simulation
void timeStepping::velVerletIntg(){
    dtSizeCalc::Parameters parDT;
    parDT.stream = stream;
    auto dtSizeCalc_ptr = std::make_shared<dtSizeCalc>(pd, nl, sys, parDT);

    fluidDynamics::Parameters parFD;
    parFD.stream = stream;
    parFD.box = box;
    parFD.rho0_f = rho0_f;
    parFD.bodyForce = bodyForce;
    parFD.c_f = c_f;
    auto fluidDynamics_ptr = std::make_shared<fluidDynamics>(pd, nl, sys, parFD);

    auto inOut_ptr = std::make_shared<inOut>(pd, nl, box);
    //the observer's coordinate
    real3 probe1Point = make_real3(std::get<0>(probe1),
                                   std::get<1>(probe1),
                                   std::get<2>(probe1));
    real3 probe2Point = make_real3(std::get<0>(probe2),
                                   std::get<1>(probe2),
                                   std::get<2>(probe2));

    //output the initial configuration
    inOut_ptr->outputToFile<inOut::VTU, inOut::Fluid>(0);
    inOut_ptr->outputToFile<inOut::VTU, inOut::Wall>(0);
    inOut_ptr->outputToFile<inOut::VTU, inOut::thirdBody>(0);

    real rcut = Kernel::getCutOff(h);
    //TODO this updateNeighbourList can be much improved
    nl->updateNeighbourList(box, rcut, stream);
    //calculate initial number density: sigma0
    fluidDynamics_ptr->calcInitNumDensity(h);

    auto t1 = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> interval;

    //computation loop starts
    while (physical_time < End_time){

        nl->updateNeighbourList(box, rcut, stream);

        fluidDynamics_ptr->densitySumFreeSurface(h);

        integration<updateOption::firstHalf>(dt);

//        fluidDynamics_ptr->densitySumFreeSurface(h);

//        fluidDynamics_ptr->densitySumLightPhase(h);

        fluidDynamics_ptr->calcDensity<fluidDynamics::densityOption::RiemannDensity>(dt, h);
        fluidDynamics_ptr->calcPressureBC(h);
        fluidDynamics_ptr->calcForce<fluidDynamics::forceOption::RiemannForce>(h, physical_time);

        integration<updateOption::secondHalf>(dt);

        dt = dtSizeCalc_ptr->calcDt(h, c_f);

        physical_time += dt;

        if (iter_counter % screen_interval == 0){
            printf("Step: %d \tTime: %0.3f \tdt: %f \n", iter_counter, physical_time, dt);
        }

        auto t2 = std::chrono::high_resolution_clock::now();
        //write results into a file
        if (output_counter < physical_time*output_interval){
            printf("|I/O| \tWriting output to disk ... file No. %d \n", output_counter+1);
            //write the simulation results into a file
            inOut_ptr->outputToFile<inOut::VTU, inOut::Fluid>(output_counter+1);
            inOut_ptr->outputToFile<inOut::VTU, inOut::thirdBody>(output_counter+1);
            //write the proble signals for a givnen obsever point into a file
            inOut_ptr->probeSignalToFile(h, probe1Point, std::get<3>(probe1), physical_time, output_counter+1);
            inOut_ptr->probeSignalToFile(h, probe2Point, std::get<3>(probe2), physical_time, output_counter+1);
            output_counter++;
        }
        auto t3 = std::chrono::high_resolution_clock::now();
        interval += t3 - t2;

        //resorting particles (slightly improves the performance)
        if(iter_counter%500 == 0){
            pd->sortParticles();
        }
        iter_counter++;
    }
    auto t4 = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> tt = t4 - t1 - interval;
    printf("Total wall clock time for computation: %.3f seconds \n", tt.count());
    printf("Total number of Iterations: %d \n", iter_counter);
}

//Dual-Criteria time integration scheme
//called from the main inerface to run the simulation
void timeStepping::dualCriteriaIntg(){
    dtSizeCalc::Parameters parDT;
    parDT.stream = stream;
    auto dtSizeCalc_ptr = std::make_shared<dtSizeCalc>(pd, nl, sys, parDT);

    fluidDynamics::Parameters parFD;
    parFD.stream = stream;
    parFD.box = box;
    parFD.rho0_f = rho0_f;
    parFD.bodyForce = bodyForce;
    parFD.c_f = c_f;
    auto fluidDynamics_ptr = std::make_shared<fluidDynamics>(pd, nl, sys, parFD);

    auto inOut_ptr = std::make_shared<inOut>(pd, nl, box);
    //output the initial configuration
    inOut_ptr->outputToFile<inOut::VTU, inOut::Fluid>(0);
    inOut_ptr->outputToFile<inOut::VTU, inOut::Wall>(0);

    real rcut = Kernel::getCutOff(h);
    //TODO this updateNeighbourList can be much improved
    nl->updateNeighbourList(box, rcut, stream);
    //calculate initial number density: sigma0
    fluidDynamics_ptr->calcInitNumDensity(h);

    //computation loop starts
    while (physical_time < End_time){

        nl->updateNeighbourList(box, rcut, stream);

        Dt = dtSizeCalc_ptr->calcDtAdv(h, U_f);
        fluidDynamics_ptr->densitySumFreeSurface(h);

        real relaxation_time = 0.0;
        while (relaxation_time < Dt){


            integration<updateOption::firstHalf>(dt);

            fluidDynamics_ptr->calcDensity<fluidDynamics::densityOption::RiemannDensity>(dt, h);
            fluidDynamics_ptr->calcPressureBC(h);
            fluidDynamics_ptr->calcForce<fluidDynamics::forceOption::RiemannForce>(h, physical_time);

            integration<updateOption::secondHalf>(dt);

            dt = dtSizeCalc_ptr->calcDtAcs(h, c_f);

            relaxation_time += dt;
            physical_time += dt;
        }

        if (iter_counter % screen_interval == 0){
            printf("Step: %d \tTime: %0.3f \tDt: %f \tdt: %f \n", iter_counter, physical_time, Dt, dt);
        }

        //write results to a file
        if (output_counter < physical_time*output_interval){
            printf("|I/O| \tWriting output to disk ... file No. %d \n", output_counter+1);
            inOut_ptr->outputToFile<inOut::VTU, inOut::Fluid>(output_counter+1);
            output_counter++;
        }

        //resorting particles
        if(iter_counter%500 == 0){
            pd->sortParticles();
        }
        iter_counter++;
    }
}

}//namspace gpu
