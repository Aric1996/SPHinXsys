#include "hip/hip_runtime.h"
/*
 * Massoud Rezavand 2019.
 * Technical University of Munich
 * gpuSPHinxsys - An SPH solver for CUDA enabled GPUs
 *
 * This module collects all the functions required for
 * the calculation of the dynamic timeStep sizes
 */

#include"dtSizeCalc.cuh"

namespace gpu{

dtSizeCalc::dtSizeCalc(shared_ptr<ParticleData> pd,
                       shared_ptr<CellList> nl,
                       shared_ptr<System> sys,
                       Parameters par):
    pd(pd), nl(nl), sys(sys),
    stream(par.stream){
    printf("|dtSizeCalc| \tis called with dtMin: %f \n", dtMin);
}

dtSizeCalc::~dtSizeCalc(){
    printf("|dtSizeCalc| \tcall ended! \n");
}

namespace dtSizeCalc_ns{

//function to get the magnitude of acc and vel
__global__ void magAccVel(int N,
                          const int* __restrict__ groupIndex,
                          real3* __restrict__ vel,
                          real4* __restrict__ force,
                          real* __restrict__ velMag,
                          real* __restrict__ AccMag){
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    if(i>=N) return;
    const real3  veli = vel[groupIndex[i]];
    const real3  acci = make_real3(force[groupIndex[i]]);
    velMag[groupIndex[i]] = sqrtf(dot(veli, veli));
    AccMag[groupIndex[i]] = sqrtf(dot(acci, acci));
}

}//dtSizeCalc_ns

//function to find the max acceleration and velocity in the systems
template<int reductionOpt>
real dtSizeCalc::maxAccVel()
{
    int np = pd->getNumParticles();
    int Nthreads=128;
    int Nblocks=np/Nthreads + ((np%Nthreads)?1:0);
    auto groupIndex = nl->getGroupIndexIterator();
#ifndef _TRANSPORT_VELOCITY_
    auto   vel = pd->getVel(access::location::gpu, access::mode::read);
#else
    auto   vel = pd->getVel_tv(access::location::gpu, access::mode::read);
#endif
    auto force = pd->getForce(access::location::gpu, access::mode::read);

    velMag.resize(np);
    AccMag.resize(np);

    auto velMag_ptr = thrust::raw_pointer_cast(velMag.data());
    auto AccMag_ptr = thrust::raw_pointer_cast(AccMag.data());

    //TODO: this coud also be improved by templates when force and vel are of the same type (e.g. real3)
    dtSizeCalc_ns::magAccVel<<<Nblocks, Nthreads, 0, stream>>>(np,
                                                               groupIndex,
                                                               vel.raw(),
                                                               force.raw(),
                                                               velMag_ptr,
                                                               AccMag_ptr);

    //find the Max of reducedPar(acc or vel) among all particles
    real *maxPar;
    hipMalloc(&maxPar, sizeof(real));
    {
        size_t newSize = 0;
        if(reductionOpt == reducedPar::Vel)
            hipcub::DeviceReduce::Max(nullptr, newSize, velMag_ptr, maxPar, np);
        else if(reductionOpt == reducedPar::Acc)
            hipcub::DeviceReduce::Max(nullptr, newSize, AccMag_ptr, maxPar, np);
        else
            throw std::runtime_error("|dtSizeCalc| \treducedPar is not valid!");


        if(newSize > tempStorage.size()){
            tempStorage.resize(newSize);
        }
    }
    size_t size = tempStorage.size();
    if(reductionOpt == reducedPar::Vel)
        hipcub::DeviceReduce::Max((void*)thrust::raw_pointer_cast(tempStorage.data()), size, velMag_ptr, maxPar, np);
    else if(reductionOpt == reducedPar::Acc)
        hipcub::DeviceReduce::Max((void*)thrust::raw_pointer_cast(tempStorage.data()), size, AccMag_ptr, maxPar, np);


    real max = 0;
    CudaSafeCall(hipMemcpy(&max, maxPar, sizeof(real), hipMemcpyDeviceToHost));
    CudaSafeCall(hipFree(maxPar));
    return max;
}

//calculate the advection time step size
//Eq. 8 in doi.org/10.1016/j.jcp.2019.109135
real dtSizeCalc::calcDtAdv(real h, real U_f){
    real cflAdv = 0.25;
    real velMax = 0.0;
    velMax = maxAccVel<reducedPar::Vel>();
    real Umax = std::max(U_f, velMax);
    //dt1 based on advection
    const real dt1 = h / (Umax + 1e-6f);
    //dt2 based on viscous terms (TODO)
    const real dt2 = std::numeric_limits<int>::max();// h*h/kinViscosity;
    //new value of the dynamic time step.
    real dtSize = cflAdv * std::min(dt1, dt2);
    if(dtSize<real(dtMin))
        dtSize=real(dtMin);
    return dtSize;
}

//calculate the acoustic time step size
//Eq. 9 in doi.org/10.1016/j.jcp.2019.109135
real dtSizeCalc::calcDtAcs(real h, real c_f){
    real cflAcs = 0.6;
    real velMax = 0.0;
    velMax = maxAccVel<reducedPar::Vel>();
    //new value of the dynamic time step.
    real dtSize = cflAcs * h / (c_f + velMax + 1e-6f);
    if(dtSize<real(dtMin))
        dtSize=real(dtMin);
    return dtSize;
}

//calculate time dynamic step size
real dtSizeCalc::calcDt(real h, real c_f){
    real CFL = 0.25;
    real accMax = 0.0;
    real velMax = 0.0;
    velMax = maxAccVel<reducedPar::Vel>();
    accMax = maxAccVel<reducedPar::Acc>();
    //dt1 based on force per unit mass.
    const real dtF = (accMax)?sqrtf(h/accMax):std::numeric_limits<int>::max();
    //dt2 based on advection
    const real dtAd = h/(std::max(c_f,velMax*10.f));
    //new value of the dynamic time step.
    real dtSize=CFL*std::min(dtF,dtAd);
    if(dtSize<real(dtMin))
        dtSize=real(dtMin);
    return dtSize;
}

}//namespace gpu
