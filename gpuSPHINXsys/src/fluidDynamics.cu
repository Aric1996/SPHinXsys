#include "hip/hip_runtime.h"
/*
 * Massoud Rezavand 2019.
 * Technical University of Munich
 * gpuSPHinxsys - An SPH solver for CUDA enabled GPUs
 *
 * This module collects all the functions required for
 * the fluid dynamics related calculations
 */

#include"fluidDynamics.cuh"
#include"Kernel.cuh"

using Kernel = gpu::KernelFunction::Wendland_C4;

namespace gpu{

fluidDynamics::fluidDynamics(shared_ptr<ParticleData> pd,
                             shared_ptr<CellList> nl,
                             shared_ptr<System> sys,
                             Parameters par):
    pd(pd), nl(nl), sys(sys),
    c_f(par.c_f), rho0_f(par.rho0_f), bodyForce(par.bodyForce),
    stream(par.stream), box(par.box){
    printf("|fluidDynamics| \tis called with c_f = %.1f, rho0_f = %.1f \n", c_f, rho0_f);
}

fluidDynamics::~fluidDynamics(){
    printf("|fluidDynamics| \tcall ended! \n");
}

namespace fluidDynamics_ns{

//Kernel to calculate drho/dt by Riemann Solvers (Continuity Eq.)
template<class NeighbourContainer, class Kernel>
__global__ void calcDensityRiemann_ker(NeighbourContainer ni,
                                       Kernel kernel,
                                       const real4* __restrict__ sortPos,
                                       const int* __restrict__ groupIndex,
                                       int np, Box box,
                                       real3* __restrict__ vel,
                                       real* __restrict__ rho,
                                       real* __restrict__ mass,
                                       real* __restrict__ p, real h, real dt,
                                       real c_f,
                                       const real* __restrict__ rho0,
                                       real* __restrict__ vol){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i >= np) return;
    //only on LIQUID particles
    const real4 posi = hipcub::ThreadLoad<hipcub::LOAD_LDG>(sortPos + i);
    if(posi.w != WALL){
        //Set ni to provide iterators for particle i
        ni.set(i);

        const real3   ri = make_real3(hipcub::ThreadLoad<hipcub::LOAD_LDG>(sortPos + i));
        const real3 veli = vel[groupIndex[i]];
        const real  rhoi = rho[groupIndex[i]];
        const real    pi = p[groupIndex[i]];
        real        drho = real();

        auto it = ni.begin(); //Iterator to the first neighbour of particle i

        while(it){
            auto neigh = *it++;
            if(neigh.getGroupIndex() == groupIndex[i]) continue; //skip if same particle

            const real3   rj = make_real3(neigh.getPos());
            const int  typej = neigh.getPos().w;
            const real3 velj = vel[neigh.getGroupIndex()];
            const real  rhoj = rho[neigh.getGroupIndex()];
            const real massj = mass[neigh.getGroupIndex()];
            const real  volj = vol[neigh.getGroupIndex()];
            const real3  rij = box.apply_pbc(ri-rj);

            //low dissipation Riemann problem
            real r2 = dot(rij, rij);
            real dist = sqrtf(r2);
            real3 _rij = rj - ri;
            real pj = p[neigh.getGroupIndex()];
            real3 e_ij = _rij*1/(dist + 1.0e-15);
            real ul = dot(e_ij, veli);
            real ur = dot(e_ij, velj);
            real v_star = (rhoi*ul+rhoj*ur+(pi-pj)/c_f)/(rhoi+rhoj);
            real aw = kernel.gradient(rij, h, box.boxSize.z);
            //only volume of wall particles into account
            if (typej == WALL)
                drho += 2.0*rhoi*volj*(v_star-ul)*aw*dist;
            else
                drho += 2.0*rhoi*massj/rhoj*(v_star-ul)*aw*dist;
        }
        rho[groupIndex[i]] += drho*dt;
        //get the volume according to rho
        vol[groupIndex[i]] = mass[groupIndex[i]]/rho[groupIndex[i]];
        // pressure calculation via the linear EoS
        p[groupIndex[i]] = c_f*c_f*(rho[groupIndex[i]] - rho0[groupIndex[i]]);
//        printf("rho = %f  and rho0 = %f \n",rho[groupIndex[i]], rho0[groupIndex[i]] );

    }
}

//Kernel to calculate drho/dt using Artificial viscosity (Continuity Eq.)
template<class NeighbourContainer, class Kernel>
__global__ void calcDensityArtificial_ker(NeighbourContainer ni,
                                          Kernel kernel,
                                          const real4* __restrict__ sortPos,
                                          const int* __restrict__ groupIndex,
                                          int np, Box box,
                                          real3* __restrict__ vel,
                                          real* __restrict__ rho,
                                          real* __restrict__ mass,
                                          real* __restrict__ p,
                                          real h, real dt,
                                          real c_f,
                                          const real* __restrict__ rho0){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i >= np) return;
    //only on LIQUID and gas particles
    const real4 posi = hipcub::ThreadLoad<hipcub::LOAD_LDG>(sortPos + i);
    if(posi.w != WALL){
        //Set ni to provide iterators for particle i
        ni.set(i);

        const real3   ri = make_real3(hipcub::ThreadLoad<hipcub::LOAD_LDG>(sortPos + i));
        const real3 veli = vel[groupIndex[i]];
        const real  rhoi = rho[groupIndex[i]];
        real        drho = real();

        auto it = ni.begin(); //Iterator to the first neighbour of particle i

        while(it){
            auto neigh = *it++;
            if(neigh.getGroupIndex() == groupIndex[i]) continue; //skip if same particle

            const real3    rj = make_real3(neigh.getPos());
            const real3  velj = vel[neigh.getGroupIndex()];
            const real   rhoj = rho[neigh.getGroupIndex()];
            const real  massj = mass[neigh.getGroupIndex()];

            const real3   rij = box.apply_pbc(ri-rj);
            //Artificial viscosity
            const real3 velij = veli - velj;
            //TODO
            //in this AV implemetaion we should use the Cubic kernel which indludes rij in there
            const real3 kernel_grad = /*rij**/rij*kernel.gradient(rij, h, box.boxSize.z);
            drho += rhoi*massj/rhoj*dot(kernel_grad, velij);
        }
        rho[groupIndex[i]] += drho*dt;
        // pressure calculation via the linear EoS
        p[groupIndex[i]] = c_f*c_f*(rho[groupIndex[i]] - rho0[groupIndex[i]]);
    }
}

//Kernel to calculate initail number density sigma0
template<class NeighbourContainer, class Kernel>
__global__ void calcInitNumDensity_ker(NeighbourContainer ni,
                                       Kernel kernel,
                                       const real4* __restrict__ sortPos,
                                       const int* __restrict__ groupIndex,
                                       real* __restrict__ sigma0,
                                       int np, Box box,
                                       real h, real rho0_f){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i >= np) return;
    ni.set(i);
    const real3 ri = make_real3(hipcub::ThreadLoad<hipcub::LOAD_LDG>(sortPos + i));
    real  sum0 = real();
    auto it = ni.begin();
    while(it){
        auto neigh = *it++;
        const real3  rj = make_real3(neigh.getPos());
        const real3 rij = box.apply_pbc(ri-rj);
        sum0 += kernel(rij, h, box.boxSize.z);
    }
    sigma0[groupIndex[i]] = sum0;
}

//Kernel to update density using summation for free surface cases
template<class NeighbourContainer, class Kernel>
__global__ void densitySumFreeSurface_ker(NeighbourContainer ni,
                                          Kernel kernel,
                                          const real4* __restrict__ sortPos,
                                          const int* __restrict__ groupIndex,
                                          real* __restrict__ sigma0,
                                          real* __restrict__ rho,
                                          int np, Box box,
                                          real h,
                                          const real* __restrict__ rho0){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i >= np) return;
    const real4 posi = hipcub::ThreadLoad<hipcub::LOAD_LDG>(sortPos + i);
    if(posi.w != WALL){
        ni.set(i);
        const real3 ri = make_real3(hipcub::ThreadLoad<hipcub::LOAD_LDG>(sortPos + i));
        const real sigma0i = sigma0[groupIndex[i]];
        real rhoi = rho[groupIndex[i]];
        real  sigma = real();
        auto it = ni.begin();
        while(it){
            auto neigh = *it++;
            const real4 posj = neigh.getPos();
            // include only Fluid neighboring particles
            if((posi.w ==LIQUID and posj.w == THIRDBODY) /*or
                    (posi.w == THIRDBODY and posj.w ==LIQUID) or
                        (posi.w == THIRDBODY and posj.w ==THIRDBODY)*/) continue;
            const real3  rj = make_real3(neigh.getPos());
            const real3 rij = box.apply_pbc(ri-rj);
            sigma += kernel(rij, h, box.boxSize.z);
        }
        real rhoSum = sigma * rho0[groupIndex[i]] / sigma0i;
        rho[groupIndex[i]] = rhoSum + fmax(0.0f, (rhoi - rhoSum)) * rho0[groupIndex[i]] / rhoi;
    }
}

//Kernel to update density using summation for the lighter phase
template<class NeighbourContainer, class Kernel>
__global__ void densitySumLightPhase_ker(NeighbourContainer ni,
                                         Kernel kernel,
                                         const real4* __restrict__ sortPos,
                                         const int* __restrict__ groupIndex,
                                         real* __restrict__ sigma0,
                                         real* __restrict__ rho,
                                         int np, Box box, real h,
                                         const real* __restrict__ rho0,
                                         real* __restrict__ p,
                                         const real c_f,
                                         real* __restrict__ vol,
                                         const real* __restrict__ mass){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i >= np) return;
    //only on gas particles
    const real4 posi = hipcub::ThreadLoad<hipcub::LOAD_LDG>(sortPos + i);
    if(posi.w == THIRDBODY){
        ni.set(i);
        const real3 ri = make_real3(hipcub::ThreadLoad<hipcub::LOAD_LDG>(sortPos + i));

        const real   rho0i = rho0[groupIndex[i]];

        real  sum = real();
        auto it = ni.begin();
        while(it){
            auto neigh = *it++;
            const real3   rj = make_real3(neigh.getPos());
            const real3  rij = box.apply_pbc(ri-rj);

            sum += kernel(rij, h, box.boxSize.z);
            /*or in a total Lagrangian form: */
            //const real rho0j = rho0[neigh.getGroupIndex()];
            //sum += kernel(rij, h, box.boxSize.z)*2.f*rho0i/(rho0i+rho0j);
        }
        rho[groupIndex[i]] = sum * mass[groupIndex[i]];
        /*or in a total Lagrangian form: */
        //const real sigma0i = sigma0[groupIndex[i]];
        //rho[groupIndex[i]] = sum * rho0i / sigma0i;

        //get the volume according to rho
        vol[groupIndex[i]] = mass[groupIndex[i]]/rho[groupIndex[i]];
        // pressure calculation via the linear EoS
        p[groupIndex[i]] = c_f*c_f*(rho[groupIndex[i]] - rho0i);
    }

}


//Kernel to calculate pressure for wall particles
template<class NeighbourContainer, class Kernel>
__global__ void calcPressureBC_ker(NeighbourContainer ni,
                                   Kernel kernel,
                                   const real4* __restrict__ sortPos,
                                   const int* __restrict__ groupIndex,
                                   int np, Box box,
                                   real* __restrict__ rho,
                                   real* __restrict__ p,
                                   real h, real c_f,
                                   const real* __restrict__ rho0,
                                   real3 bodyForce){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i >= np) return;
    //only on Wall particles
    const real4 posi = hipcub::ThreadLoad<hipcub::LOAD_LDG>(sortPos + i);
    if(posi.w == WALL){
        //Set ni to provide iterators for particle i
        ni.set(i);

        const real3 ri = make_real3(hipcub::ThreadLoad<hipcub::LOAD_LDG>(sortPos + i));
        real  sum0 = real();
        real  sum1 = real();
        real3 sum2 = real3();
        real3   aw = real3();
        //gravity
        //in case coordinates are needed ow. directly can be used
        const real3 body_force = bodyForce;

        auto it = ni.begin(); //Iterator to the first neighbour of particle i

        while(it){
            auto neigh = *it++;
            if(neigh.getGroupIndex() == groupIndex[i]) continue; //skip if same particle
            const real4 posj = neigh.getPos();
            // include only Fluid neighboring particles
            if(posj.w == WALL) continue;

            const real3  rj = make_real3(neigh.getPos());
            const real rhoj = rho[neigh.getGroupIndex()];
            const real   pj = p[neigh.getGroupIndex()];
            const real3 rij = box.apply_pbc(ri-rj);
            const real  wij = kernel(rij, h, box.boxSize.z);
//            printf("rhoj gas = %f  \trho0j gas = %f \n", rho[neigh.getGroupIndex()], rho0[neigh.getGroupIndex()]);

            //fraction devided by rho_j to get pressure from the lighter fluid
            sum0 += wij/rhoj;
            sum1 += pj*wij/rhoj;
            sum2 += rij*wij*rhoj/rhoj;

        }
        aw = body_force;// - aw; //for later developments
        real tmp = real();
        tmp = dot(aw, sum2);
        p[groupIndex[i]] = (sum1+tmp)/(sum0+1.e-20);
        //get density for wall particles
        rho[groupIndex[i]] = p[groupIndex[i]]/(c_f*c_f) + rho0[groupIndex[i]];
//        printf("rho gas = %f  \trho0 gas = %f \n", rho[groupIndex[i]], rho0[groupIndex[i]]);
    }
}

//Kernel to calculate pressure and viscosity related forces via RiemannSolvers
template<class NeighbourContainer, class Kernel>
__global__ void calcForceRiemann_ker(NeighbourContainer ni,
                                     Kernel kernel,
                                     const real4* __restrict__ sortPos,
                                     const int* __restrict__ groupIndex,
                                     int np, Box box,
                                     real4* __restrict__ force,
                                     real3* __restrict__ vel,
                                     real* __restrict__ rho,
                                     real* __restrict__ mass,
                                     real* __restrict__ p,
                                     real3* __restrict__ vel_tv,
                                     real3* __restrict__ F_Pb, real P_b,
                                     real h, real c_f, real3 bodyForce,
                                     real physicalTime,
                                     real* __restrict__ vol){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i >= np) return;
    //only on Fluid particles
    const real4 posi = hipcub::ThreadLoad<hipcub::LOAD_LDG>(sortPos + i);
    if(posi.w != WALL){
        //gravity
#ifdef _TIMEDEPENDENT_BODYFORCE_    //for sloshing tank
         const real4 body_force = make_real4(bodyForce.x*sin(2.*M_PI*0.496*physicalTime)
                                             ,bodyForce.y, 0., 0.);
#else
         const real4 body_force = make_real4(bodyForce, 0.);
#endif

        //Set ni to provide iterators for particle i
        ni.set(i);

        const real3   ri = make_real3(hipcub::ThreadLoad<hipcub::LOAD_LDG>(sortPos + i));
        const real  rhoi = rho[groupIndex[i]];
        const real    pi = p[groupIndex[i]];
        const real3 veli = vel[groupIndex[i]];
        real3 F1 = real3();

        auto it = ni.begin();

        while(it){
            auto neigh = *it++;

            if(neigh.getGroupIndex() == groupIndex[i]) continue; //skip if same particle

            const real3   rj = make_real3(neigh.getPos());
            const real  rhoj = rho[neigh.getGroupIndex()];
            const real    pj = p[neigh.getGroupIndex()];
            const real massj = mass[neigh.getGroupIndex()];
            const int  typej = neigh.getPos().w;
            real volj;
            //only volume of wall particles into account
            if(typej != WALL)
                volj = massj/rhoj;
            else
                volj = vol[neigh.getGroupIndex()];
            const real3 velj = vel[neigh.getGroupIndex()];
            const real3  rij = box.apply_pbc(ri-rj);
            //low dissipation Riemann problem
            const real3 kernel_grad = rij*kernel.gradient(rij, h, box.boxSize.z);
            const real4 posj = neigh.getPos();
            if(posj.w != WALL){
                real     r2 = dot(rij, rij);
                real   dist = sqrtf(r2);
                real3  _rij = rj - ri;
                real3  e_ij = _rij*1/(dist + 1.0e-15);
                real     ul = dot(e_ij, veli);
                real     ur = dot(e_ij, velj);
                real p_star = (rhoi*pj+rhoj*pi+rhoi*rhoj*c_f*(ul-ur)*
                               fmin(real(3.0)*fmax((ul-ur)/c_f, real(0.0)), real(1.0)))/(rhoi+rhoj);
                real  temp1 = -2.0*p_star*volj/rhoi;
                F1 += temp1*kernel_grad;
            }else{
                //exclude the second term only when Fluid-Wall interaction
                real p_star = (rhoj*pi + rhoi*pj)/(rhoi + rhoj);
                real temp1 = -2.0*p_star*volj/rhoi;
                F1 += temp1*kernel_grad;
            }
            //transport velocity formulation
#ifdef _TRANSPORT_VELOCITY_
            if(posi.w == THIRDBODY){
                const real3 v_tv_i = vel_tv[groupIndex[i]];
                const real3 v_tv_j = vel_tv[neigh.getGroupIndex()];
                const real   massi = mass[groupIndex[i]];
                const real    voli = massi/rhoi;
                const real    coef = real(1.)/massi*(voli*voli + volj*volj);
                // artificial stress tensor Aij (Adami et al. 2013)
                real3 A_ij = real3();
                // x component
                real3 Ax_i = (v_tv_i - veli) * rhoi * veli.x;
                real3 Ax_j = (v_tv_j - velj) * rhoj * velj.x;
                A_ij.x = real(0.5)*dot((Ax_i+Ax_j), kernel_grad);
                // y component
                real3 Ay_i = (v_tv_i - veli) * rhoi * veli.y;
                real3 Ay_j = (v_tv_j - velj) * rhoj * velj.y;
                A_ij.y = real(0.5)*dot((Ay_i+Ay_j), kernel_grad);
                // z component
                real3 Az_i = (v_tv_i - veli) * rhoi * veli.z;
                real3 Az_j = (v_tv_j - velj) * rhoj * velj.z;
                A_ij.z = real(0.5)*dot((Az_i+Az_j), kernel_grad);

                real3 dF_AS = A_ij * coef;
                F1 += dF_AS;
                // background pressure force
                real P_b1 = 5.*0.001*c_f*c_f;
                real     temp3 = real(-2.)*P_b1*volj/rhoi;
                real3  dF_Pb = kernel_grad * temp3;
                F_Pb[groupIndex[i]] += dF_Pb;
            }
#endif
        }
        force[groupIndex[i]] = make_real4(F1, 0);
//        if(posi.w == LIQUID)
        force[groupIndex[i]] += body_force;
    }
}

//Kernel to calculate pressure and Artificial viscosity forces
template<class NeighbourContainer, class Kernel>
__global__ void calcForceArtificial_ker(NeighbourContainer ni,
                                        Kernel kernel,
                                        const real4* __restrict__ sortPos,
                                        const int* __restrict__ groupIndex,
                                        int np, Box box,
                                        real4* __restrict__ force,
                                        real3* __restrict__ vel,
                                        real* __restrict__ rho,
                                        real* __restrict__ mass,
                                        real* __restrict__ p,
                                        real h, real c_f, real3 bodyForce){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i >= np) return;
    //only on Fluid particles
    const real4 posi = hipcub::ThreadLoad<hipcub::LOAD_LDG>(sortPos + i);
    if(posi.w == LIQUID){
        //gravity
        const real4 body_force = make_real4(bodyForce, 0.);

        //Set ni to provide iterators for particle i
        ni.set(i);

        const real3   ri = make_real3(hipcub::ThreadLoad<hipcub::LOAD_LDG>(sortPos + i));
        const real  rhoi = rho[groupIndex[i]];
        const real    pi = p[groupIndex[i]];
        const real massi = mass[groupIndex[i]];
        const real  voli = massi/rhoi;
        const real3 veli = vel[groupIndex[i]];

        real3 F1 = real3();
        real3 F2 = real3();

        auto it = ni.begin();

        while(it){
            auto neigh = *it++;

            if(neigh.getGroupIndex() == groupIndex[i]) continue; //skip if same particle

            const real3   rj = make_real3(neigh.getPos());
            const real  rhoj = rho[neigh.getGroupIndex()];
            const real    pj = p[neigh.getGroupIndex()];
            const real massj = mass[neigh.getGroupIndex()];
            const real  volj = massj/rhoj;
            const real3 velj = vel[neigh.getGroupIndex()];
            const real3  rij = box.apply_pbc(ri-rj);
            //Artificial viscosity
            //TODO
            //in this AV implemetaion we should use the Cubic kernel which indludes rij in there
            const real3 kernel_grad = /*rij**/rij*kernel.gradient(rij, h, box.boxSize.z);
            const real temp0 = 1./massi*(voli*voli + volj*volj);
            const real   pij = (rhoj*pi + rhoi*pj)/(rhoi + rhoj);
            const real temp1 = -1.*pij*temp0;
            F1 += temp1*kernel_grad;

            const real4   posj = neigh.getPos();
            //TODO
            //if(posj.w == LIQUID){ //free-slip
            const real   alpha = 0.1;
            const real epsilon = 0.001;
            const real   rhoij = (rhoi + rhoj)/2.;
            const real3  velij = veli - velj;
            const real  vij_dr = dot(velij, rij)/(dot(rij, rij)+epsilon*h*h);
            const real    visc = -massj*alpha*c_f*h*vij_dr/rhoij;
            F2 += visc*kernel_grad;
            //}
        }
        force[groupIndex[i]] = make_real4(F1+F2, 0);
        force[groupIndex[i]] += body_force;
    }
}

}//namspace fluidDynamics_ns


//calculate density
template<int densityOpt>
void fluidDynamics::calcDensity(real Dt, real h){
    int np = pd->getNumParticles();
    Kernel kernel;
    // get a NeighborContainer
    auto ni = nl->getNeighbourContainer();
    auto sortPos = nl->getPositionIterator();
    auto groupIndex = nl->getGroupIndexIterator();
    auto vel = pd->getVel(access::location::gpu, access::mode::readwrite).raw();
    //If mass is not allocated assume all masses are 1
    real *mass = nullptr;
    if(pd->isMassAllocated())
        mass = pd->getMass(access::location::gpu, access::mode::read).raw();
    auto rho = pd->getRho(access::location::gpu, access::mode::readwrite).raw();
    auto rho0 = pd->getRho0(access::location::gpu, access::mode::readwrite).raw();
    auto pressure = pd->getPressure(access::location::gpu, access::mode::readwrite).raw();
    auto vol = pd->getVol(access::location::gpu, access::mode::readwrite).raw();

    if(densityOpt==densityOption::RiemannDensity){
        fluidDynamics_ns::calcDensityRiemann_ker<<<np/128+1, 128, 0, stream>>>(ni, kernel,
                                                                               sortPos, groupIndex,
                                                                               np, box,
                                                                               vel, rho,
                                                                               mass, pressure,
                                                                               h, Dt, c_f, rho0, vol);
    }else if(densityOpt==densityOption::Continuity){
        fluidDynamics_ns::calcDensityArtificial_ker<<<np/128+1, 128, 0, stream>>>(ni, kernel,
                                                                                  sortPos, groupIndex,
                                                                                  np, box,
                                                                                  vel, rho,
                                                                                  mass, pressure,
                                                                                  h, Dt, c_f, rho0);
    }else {
        throw std::runtime_error("|fluidDynamics| \tdensityOpt is not valid!");
    }
}

//calculate initial number density: sigma0
void fluidDynamics::calcInitNumDensity(real h){
    int np = pd->getNumParticles();
    Kernel kernel;
    // get a NeighborContainer
    auto ni = nl->getNeighbourContainer();
    auto sortPos = nl->getPositionIterator();
    auto groupIndex = nl->getGroupIndexIterator();
    auto sigma0 = pd->getSigma0(access::location::gpu, access::mode::readwrite).raw();

    fluidDynamics_ns::calcInitNumDensity_ker<<<np/128+1, 128, 0, stream>>>(ni, kernel, sortPos,
                                                                           groupIndex, sigma0,
                                                                           np, box, h, rho0_f);
}

//density calculation using summation for free surface cases
void fluidDynamics::densitySumFreeSurface(real h){
    int np = pd->getNumParticles();
    Kernel kernel;
    // get a NeighborContainer
    auto ni = nl->getNeighbourContainer();
    auto sortPos = nl->getPositionIterator();
    auto groupIndex = nl->getGroupIndexIterator();
    auto sigma0 = pd->getSigma0(access::location::gpu, access::mode::readwrite).raw();
    auto density = pd->getRho(access::location::gpu, access::mode::readwrite).raw();
    auto rho0 = pd->getRho0(access::location::gpu, access::mode::readwrite).raw();

    fluidDynamics_ns::densitySumFreeSurface_ker<<<np/128+1, 128, 0, stream>>>(ni, kernel, sortPos,
                                                                              groupIndex, sigma0,
                                                                              density,
                                                                              np, box,
                                                                              h, rho0);
}

//density calculation using summation for gas phase
void fluidDynamics::densitySumLightPhase(real h){
    int np = pd->getNumParticles();
    Kernel kernel;
    // get a NeighborContainer
    auto ni = nl->getNeighbourContainer();
    auto sortPos = nl->getPositionIterator();
    auto groupIndex = nl->getGroupIndexIterator();
    auto sigma0 = pd->getSigma0(access::location::gpu, access::mode::readwrite).raw();
    auto density = pd->getRho(access::location::gpu, access::mode::readwrite).raw();
    auto rho0 = pd->getRho0(access::location::gpu, access::mode::readwrite).raw();
    auto pressure = pd->getPressure(access::location::gpu, access::mode::readwrite).raw();
    auto vol = pd->getVol(access::location::gpu, access::mode::readwrite).raw();
    auto mass = pd->getMass(access::location::gpu, access::mode::readwrite).raw();

    fluidDynamics_ns::densitySumLightPhase_ker<<<np/128+1, 128, 0, stream>>>(ni, kernel, sortPos,
                                                                             groupIndex, sigma0,
                                                                             density,
                                                                             np, box,
                                                                             h, rho0,
                                                                             pressure, c_f,
                                                                             vol, mass);
}

//calculate pressure for wall particles
void fluidDynamics::calcPressureBC(real h){
    int np = pd->getNumParticles();
    Kernel kernel;
    // get a NeighborContainer
    auto ni = nl->getNeighbourContainer();
    auto sortPos = nl->getPositionIterator();
    auto groupIndex = nl->getGroupIndexIterator();
    auto d_density = pd->getRho(access::location::gpu, access::mode::readwrite).raw();
    auto pressure = pd->getPressure(access::location::gpu, access::mode::readwrite).raw();
    auto rho0 = pd->getRho0(access::location::gpu, access::mode::readwrite).raw();

    fluidDynamics_ns::calcPressureBC_ker<<<np/128+1, 128, 0, stream>>>(ni, kernel, sortPos,
                                                                       groupIndex, np, box,
                                                                       d_density, pressure,
                                                                       h, c_f, rho0, bodyForce);
}

//calculate forces
template<int forceOpt>
void fluidDynamics::calcForce(real h, real physicalTime){
    int np = pd->getNumParticles();
    Kernel kernel;
    // get a NeighborContainer
    auto ni = nl->getNeighbourContainer();
    auto sortPos = nl->getPositionIterator();
    auto groupIndex = nl->getGroupIndexIterator();
    auto vel = pd->getVel(access::location::gpu, access::mode::readwrite).raw();
    //If mass is not allocated assume all masses are 1
    real *d_mass = nullptr;
    if(pd->isMassAllocated())
        d_mass = pd->getMass(access::location::gpu, access::mode::read).raw();
    auto rho = pd->getRho(access::location::gpu, access::mode::readwrite).raw();
    auto pressure = pd->getPressure(access::location::gpu, access::mode::readwrite).raw();
    auto force = pd->getForce(access::location::gpu, access::mode::readwrite).raw();
    auto vol = pd->getVol(access::location::gpu, access::mode::readwrite).raw();
#ifdef _TRANSPORT_VELOCITY_
    real    P_b = 5.0f*1.f*c_f*c_f; //the background pressure
    auto vel_tv = pd->getVel_tv(access::location::gpu, access::mode::readwrite).raw();
    auto   F_Pb = pd->getF_Pb(access::location::gpu, access::mode::readwrite).raw();
#else
    real    P_b = 0.0f;
    auto vel_tv = nullptr;
    auto   F_Pb = nullptr;
#endif

    if(forceOpt==forceOption::RiemannForce){
        fluidDynamics_ns::calcForceRiemann_ker<<<np/128+1, 128, 0, stream>>>(ni, kernel, sortPos,
                                                                             groupIndex, np, box,
                                                                             force, vel,
                                                                             rho, d_mass,
                                                                             pressure,
                                                                             vel_tv,
                                                                             F_Pb, P_b,
                                                                             h, c_f, bodyForce,
                                                                             physicalTime,
                                                                             vol);
    }else if(forceOpt==forceOption::Artificial){
        fluidDynamics_ns::calcForceArtificial_ker<<<np/128+1, 128, 0, stream>>>(ni, kernel, sortPos,
                                                                                groupIndex, np, box,
                                                                                force, vel,
                                                                                rho, d_mass,
                                                                                pressure,
                                                                                h, c_f, bodyForce);
    }else {
        throw std::runtime_error("|fluidDynamics| \tforceOpt is not valid!");
    }
}


}//namespace gpu
