#include "hip/hip_runtime.h"
/*
 * Massoud Rezavand 2019.
 * Technical University of Munich
 * gpuSPHinxsys - An SPH solver for CUDA enabled GPUs
 *
 * This module writes the particle data on an external file
 * for post-processing porpuses
 * Two output formats are available, VTU as well as PLT
 */

#include"inOut.cuh"
#include"Kernel.cuh"

using Kernel = gpu::KernelFunction::Wendland_C4;

namespace gpu{

inOut::inOut(shared_ptr<ParticleData> pd,
             shared_ptr<CellList> nl,
             Box box):
    pd(pd), nl(nl){
    printf("|inOut| \tis called  \n");
    if (fs::exists(output_folder) || fs::exists(observer_folder)){
        fs::remove_all(output_folder);
        fs::remove_all(observer_folder);
    }
    if (!fs::exists(output_folder) || !fs::exists(observer_folder)){
        fs::create_directory(output_folder);
        fs::create_directory(observer_folder);
        printf("|inOut| \toutput folders created!  \n");
    }
}

inOut::~inOut(){
    printf("|inOut| \tcall ended! \n");
}

namespace inOut_ns{

//kernel to get the physical signal at the probe
template<class NeighbourContainer>
__global__ void calcPressureSignal(NeighbourContainer ni,
                                   Box box,
                                   real h,
                                   Kernel kernel,
                                   real3 probe,
                                   real* __restrict__ probeSignal,
                                   real* __restrict__ mass,
                                   real* __restrict__ rho,
                                   real* __restrict__ p){
    real sum0 = real();
    real sum1 = real();
    auto   it = ni.begin(probe);

    while(it){
        auto neigh = *it++;
        const real4 posj = neigh.getPos();
        // include only Fluid neighboring particles
        if(posj.w == WALL) continue;
        const real3   rj = make_real3(neigh.getPos());
        const real  rhoj = rho[neigh.getGroupIndex()];
        const real    pj = p[neigh.getGroupIndex()];
        const real massj = mass[neigh.getGroupIndex()];
        const real3  rij = box.apply_pbc(probe-rj);
        sum0 += kernel(rij, h, box.boxSize.z)*massj/rhoj;
        sum1 += pj*kernel(rij, h, box.boxSize.z)*massj/rhoj;
    }
    *probeSignal = sum1/fmax(sum0, Eps);
}

}//inOut_ns


template<int outputOpt, int bodyOpt>
inline void inOut::outputToFile(int outputCount){

    const int         np = pd->getNumParticles();
    const real4 *posType = pd->getPos(access::location::cpu, access::mode::read).raw();
    const real3     *vel = pd->getVel(access::location::cpu, access::mode::read).raw();
    const real      *rho = pd->getRho(access::location::cpu, access::mode::read).raw();
    const real    *press = pd->getPressure(access::location::cpu, access::mode::read).raw();

    //seperate particles of different types
    if(bodyOpt==Fluid){
        posTypeFluid.clear();
        velFluid.clear();
        rhoFluid.clear();
        pressFluid.clear();
        for (size_t i = 0; i < np; i++){
            if (posType[i].w==LIQUID){
                posTypeFluid.push_back(posType[i]);
                velFluid.push_back(vel[i]);
                rhoFluid.push_back(rho[i]);
                pressFluid.push_back(press[i]);
            }
        }
        posTypeBody = posTypeFluid;
        velBody = velFluid;
        rhoBody = rhoFluid;
        pressBody = pressFluid;
        bodyName = "Fluid_";
        npBody = posTypeBody.size();
    }else if(bodyOpt==Wall){
        posTypeWall.clear();
        velWall.clear();
        rhoWall.clear();
        pressWall.clear();
        for (size_t i = 0; i < np; i++){
            if (posType[i].w==WALL){
                posTypeWall.push_back(posType[i]);
                velWall.push_back(vel[i]);
                rhoWall.push_back(rho[i]);
                pressWall.push_back(press[i]);
            }
        }
        posTypeBody = posTypeWall;
        velBody = velWall;
        rhoBody = rhoWall;
        pressBody = pressWall;
        bodyName = "Wall_";
        npBody = posTypeBody.size();
    }else if(bodyOpt==thirdBody){
        posTypeThirdBody.clear();
        velThirdBody.clear();
        rhoThirdBody.clear();
        pressThirdBody.clear();
        for (size_t i = 0; i < np; i++){
            if (posType[i].w==THIRDBODY){
                posTypeThirdBody.push_back(posType[i]);
                velThirdBody.push_back(vel[i]);
                rhoThirdBody.push_back(rho[i]);
                pressThirdBody.push_back(press[i]);
            }
        }
        posTypeBody = posTypeThirdBody;
        velBody = velThirdBody;
        rhoBody = rhoThirdBody;
        pressBody = pressThirdBody;
        bodyName = "ThirdBody_";
        npBody = posTypeBody.size();
    }else{
        throw std::runtime_error("|inOut| \tBody option to output is not valid!");
    }

    // write in VTU format to use in Paraview
    if(outputOpt==VTU)
    {
        std::string filefullpath = output_folder + "/" + bodyName + std::to_string(outputCount) + ".vtu";
        std::ofstream out(filefullpath.c_str(), std::ios::trunc);

        //beginning of the XML file
        out << "<?xml version=\"1.0\"?>\n";
        out << "<VTKFile type=\"UnstructuredGrid\" version=\"0.1\" byte_order=\"LittleEndian\">\n";
        out << " <UnstructuredGrid>\n";
        out << "  <Piece Name =\"" << 0 << "\" NumberOfPoints=\"" << npBody << "\" NumberOfCells=\"0\">\n";

        //write position of particles
        out << "   <Points>\n";
        out << "    <DataArray Name=\"Position\" type=\"Float32\"  NumberOfComponents=\"3\" Format=\"ascii\">\n";
        out << "    ";
        for (size_t i = 0; i < npBody; i++) {
            out << posTypeBody[i].x << " " << posTypeBody[i].y << " " << posTypeBody[i].z << " ";
        }
        out << std::endl;
        out << "    </DataArray>\n";
        out << "   </Points>\n";

        //Particles data set
        out << "   <PointData  Vectors=\"vector\">\n";
        //wrtie density
        out << "    <DataArray Name=\"Density\" type=\"Float32\" Format=\"ascii\">\n";
        out << "    ";
        for (size_t i = 0; i < npBody; i++) {
            out << rhoBody[i] << " ";
        }
        out << std::endl;
        out << "    </DataArray>\n";

        //wrtie type
        out << "    <DataArray Name=\"Type\" type=\"Int32\" Format=\"ascii\">\n";
        out << "    ";
        for (size_t i = 0; i < npBody; i++) {
            out << posTypeBody[i].w << " ";
        }
        out << std::endl;
        out << "    </DataArray>\n";

        //wrtie id
        out << "    <DataArray Name=\"Id\" type=\"Int32\" Format=\"ascii\">\n";
        out << "    ";
        for (size_t i = 0; i < npBody; i++) {
            //            out << id[i] << " ";
            out << i << " ";
        }
        out << std::endl;
        out << "    </DataArray>\n";

        //write pressure
        out << "    <DataArray Name=\"Pressure\" type=\"Float32\" Format=\"ascii\">\n";
        out << "    ";
        for (size_t i = 0; i < npBody; i++) {
            out << pressBody[i] << " ";
        }
        out << std::endl;
        out << "    </DataArray>\n";

        //write velocity
        out << "    <DataArray Name=\"Velocity\" type=\"Float32\"  NumberOfComponents=\"3\" Format=\"ascii\">\n";
        out << "    ";
        for (size_t i = 0; i < npBody; i++) {
            out << velBody[i].x << " " << velBody[i].y << " " << velBody[i].z << " ";
        }
        out << std::endl;
        out << "    </DataArray>\n";

        //Particles data set ended
        out << "   </PointData>\n";

        //cells connectivity
        out << "   <Cells>\n";
        out << "    <DataArray type=\"Int32\"  Name=\"connectivity\"  Format=\"ascii\">\n";
        out << "    </DataArray>\n";
        out << "    <DataArray type=\"Int32\"  Name=\"offsets\"  Format=\"ascii\">\n";
        out << "    </DataArray>\n";
        out << "    <DataArray type=\"types\"  Name=\"offsets\"  Format=\"ascii\">\n";
        out << "    </DataArray>\n";
        out << "   </Cells>\n";

        out << "  </Piece>\n";
        out << " </UnstructuredGrid>\n";
        out << "</VTKFile>\n";

        out.close();
    }

    // write in PLT format to use in TecPlot
    else if (outputOpt==PLT) {
        std::string filefullpath = output_folder + "/" + bodyName + std::to_string(outputCount) + ".plt";
        std::ofstream out(filefullpath.c_str(), std::ios::trunc);

        out<<"VARIABLES = \"x\",\"y\",\"z\",\"type\",\"vx\",\"vy\",\"vz\",\"rho\",\"p\"\n";
        for (int i = 0; i < npBody; i++){
            out << posTypeBody[i].x << " "
                << posTypeBody[i].y << " "
                << posTypeBody[i].z << " "
                << posTypeBody[i].w << " "
                << velBody[i].x << " " << velBody[i].y << " " << velBody[i].z << " "
                << rhoBody[i] << "  "
                << pressBody[i] << "\n";
        }
        out.close();
    }

    else
    {
        throw std::runtime_error("|inOut| \tOutput format is not valid!");
    }
}

//function to output the probe signal at a given probe point into a plt file
inline void inOut::probeSignalToFile(real h, real3 probe, std::string probeID, real physicalTime, int counter){

    Kernel kernel;

    real *probeSignalTmp;
    hipMalloc(&probeSignalTmp, sizeof(real));

    // get a NeighborContainer
    auto ni = nl->getNeighbourContainer();

    real *mass = nullptr;
    if(pd->isMassAllocated())
        mass = pd->getMass(access::location::gpu, access::mode::read).raw();
    auto density = pd->getRho(access::location::gpu, access::mode::read).raw();
    auto pressure = pd->getPressure(access::location::gpu, access::mode::read).raw();

    inOut_ns::calcPressureSignal<<<1, 1>>>(ni, box, h, kernel, probe,
                                           probeSignalTmp, mass, density, pressure);

    real probeSignal = real(0.0);
    CudaSafeCall(hipMemcpy(&probeSignal, probeSignalTmp, sizeof(real), hipMemcpyDeviceToHost));
    CudaSafeCall(hipFree(probeSignalTmp));

    std::string filefullpath = observer_folder + "/" + probeID + ".plt";
    std::ofstream out(filefullpath.c_str(), std::ios::app);

    if (counter == 1){
        out<<"VARIABLES=\"time\",\"probeSignal\"\n";
    }
    out <<physicalTime<<"  "<<probeSignal<<"\n";
}


}//namespace gpu
